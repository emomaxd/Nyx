#include "hip/hip_runtime.h"
#include "nbody_cuda.cuh"
#include <iostream>
#include <cmath>

NBodySimulation::NBodySimulation(size_t num_Particles)
    : num_Particles(num_Particles)
{
    h_Positions.resize(num_Particles);
    h_Velocities.resize(num_Particles);
    h_Accelerations.resize(num_Particles);
    h_Masses.resize(num_Particles);
    h_Radii.resize(num_Particles);
    h_Densities.resize(num_Particles);

    hipMalloc(&d_Positions, num_Particles * sizeof(float3));
    hipMalloc(&d_Velocities, num_Particles * sizeof(float3));
    hipMalloc(&d_Accelerations, num_Particles * sizeof(float3));
    hipMalloc(&d_Masses, num_Particles * sizeof(float));
    hipMalloc(&d_Radii, num_Particles * sizeof(float));
    hipMalloc(&d_Densities, num_Particles * sizeof(float));

    initializeParticles();
    transferDataToDevice();
}

NBodySimulation::~NBodySimulation() {
    hipFree(d_Positions);
    hipFree(d_Velocities);
    hipFree(d_Accelerations);
    hipFree(d_Masses);
    hipFree(d_Radii);
    hipFree(d_Densities);
}

void NBodySimulation::initializeParticles() {
    std::mt19937 gen(static_cast<unsigned>(time(nullptr)));
    std::uniform_real_distribution<float> posDistX(20.0f, 1400.0f);
    std::uniform_real_distribution<float> posDistY(10.0f, 900.0f);
    std::uniform_real_distribution<float> posDistZ(-10000.0f, 10000.0f);
    std::uniform_real_distribution<float> massDist(1.0f, 1e7f);
    std::uniform_real_distribution<float> radiusDist(0.1f, 5.0f);

    for (uint32_t i = 0; i < num_Particles; ++i) {
        h_Positions[i] = make_float3(posDistX(gen), posDistY(gen), posDistZ(gen));
        h_Velocities[i] = make_float3(0.0f, 0.0f, 0.0f);
        h_Accelerations[i] = make_float3(0.0f, 0.0f, 0.0f);
        h_Masses[i] = massDist(gen);
        h_Radii[i] = radiusDist(gen);
        h_Densities[i] = h_Masses[i] / (4.0f / 3.0f * M_PI * std::pow(h_Radii[i], 3));
    }
}

void NBodySimulation::transferDataToDevice() {
    hipMemcpy(d_Positions, h_Positions.data(), num_Particles * sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(d_Velocities, h_Velocities.data(), num_Particles * sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(d_Accelerations, h_Accelerations.data(), num_Particles * sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(d_Masses, h_Masses.data(), num_Particles * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Radii, h_Radii.data(), num_Particles * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Densities, h_Densities.data(), num_Particles * sizeof(float), hipMemcpyHostToDevice);
}

void NBodySimulation::transferDataToHost() {
    hipMemcpy(h_Positions.data(), d_Positions, num_Particles * sizeof(float3), hipMemcpyDeviceToHost);
    hipMemcpy(h_Velocities.data(), d_Velocities, num_Particles * sizeof(float3), hipMemcpyDeviceToHost);
    hipMemcpy(h_Accelerations.data(), d_Accelerations, num_Particles * sizeof(float3), hipMemcpyDeviceToHost);
}

void NBodySimulation::update(Timestep ts) {

    int threadsPerBlock = 256;
    int blocks = (num_Particles + threadsPerBlock - 1) / threadsPerBlock;

    nBodyKernel<<<blocks, threadsPerBlock>>>(d_Positions, d_Velocities, d_Accelerations,
                                             d_Masses, d_Radii, d_Densities,
                                             num_Particles, ts);

    hipDeviceSynchronize();

    transferDataToHost();
}

hipError_t NBodySimulation::checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        std::cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
        std::exit(EXIT_FAILURE);
    }
    return result;
}

__global__ void nBodyKernel(float3* d_Positions, float3* d_Velocities, float3* d_Accelerations,
                             float* d_Masses, float* d_Radii, float* d_Densities,
                             uint32_t num_Particles, float dt) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_Particles) return;

    const float G = 6.67408f * 1e-11f;
    const float maxSpeed = 2000.0f; // Maximum allowed speed

    float3 position = d_Positions[idx];
    float3 velocity = d_Velocities[idx];
    float3 acceleration = make_float3(0.0f, 0.0f, 0.0f);

    // Compute gravitational forces
    for (uint32_t i = 0; i < num_Particles; ++i) {
        if (i == idx) continue;

        float3 otherPosition = d_Positions[i];
        float3 direction = make_float3(otherPosition.x - position.x,
                                       otherPosition.y - position.y,
                                       otherPosition.z - position.z);

        float distSq = direction.x * direction.x + direction.y * direction.y + direction.z * direction.z;
        float dist = sqrtf(distSq + 1e-7f); // Avoid division by zero
        float force = G * (d_Masses[i] * d_Masses[idx]) / (distSq + 1e-7f);

        direction.x /= dist;
        direction.y /= dist;
        direction.z /= dist;

        acceleration.x += force * direction.x;
        acceleration.y += force * direction.y;
        acceleration.z += force * direction.z;
    }

    // Update velocities and positions
    velocity.x += acceleration.x * dt;
    velocity.y += acceleration.y * dt;
    velocity.z += acceleration.z * dt;

    position.x += velocity.x * dt;
    position.y += velocity.y * dt;
    position.z += velocity.z * dt;

    // Clamp positions to bounds
    position.x = fminf(fmaxf(position.x, 0.0f), 1600.0f);
    position.y = fminf(fmaxf(position.y, 0.0f), 900.0f);
    position.z = fminf(fmaxf(position.z, -10000.0f), 10000.0f);

    // Limit speed to maxSpeed
    float speedSq = velocity.x * velocity.x + velocity.y * velocity.y + velocity.z * velocity.z;
    if (speedSq > maxSpeed * maxSpeed) {
        float scale = maxSpeed / sqrtf(speedSq);
        velocity.x *= scale;
        velocity.y *= scale;
        velocity.z *= scale;
    }

    // Write back updated values
    d_Positions[idx] = position;
    d_Velocities[idx] = velocity;
    d_Accelerations[idx] = acceleration; // Update acceleration for the next step
}

