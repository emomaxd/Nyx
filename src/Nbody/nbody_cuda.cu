#include "hip/hip_runtime.h"
#include "nbody_cuda.cuh"
#include <stdio.h>
#include <math.h>
#include <signal.h>
#include <stdlib.h>
#include <algorithm>

#define BOX_SIZE 100.0f  // Box dimension
#define TIME_STEP 0.01f  // Time step for simulation

volatile sig_atomic_t stop_simulation = 0;

__global__ void update_positions(Sphere *spheres, int num_spheres, float dt) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_spheres) return;

    Sphere &s = spheres[idx];
    
    // Update position based on velocity
    s.position.x += s.velocity.x * dt;
    s.position.y += s.velocity.y * dt;
    s.position.z += s.velocity.z * dt;

    // Handle collisions with box boundaries
    if (s.position.x - s.radius < 0.0f || s.position.x + s.radius > BOX_SIZE) {
        s.velocity.x *= -1.0f;
        s.position.x = fminf(fmaxf(s.position.x, s.radius), BOX_SIZE - s.radius);
    }
    if (s.position.y - s.radius < 0.0f || s.position.y + s.radius > BOX_SIZE) {
        s.velocity.y *= -1.0f;
        s.position.y = fminf(fmaxf(s.position.y, s.radius), BOX_SIZE - s.radius);
    }
    if (s.position.z - s.radius < 0.0f || s.position.z + s.radius > BOX_SIZE) {
        s.velocity.z *= -1.0f;
        s.position.z = fminf(fmaxf(s.position.z, s.radius), BOX_SIZE - s.radius);
    }
}

__global__ void resolve_collisions(Sphere *spheres, int num_spheres) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_spheres) return;

    Sphere &s1 = spheres[idx];

    for (int j = idx + 1; j < num_spheres; j++) {
        Sphere &s2 = spheres[j];

        // Calculate the vector between the two spheres
        float3 diff = make_float3(s2.position.x - s1.position.x,
                                  s2.position.y - s1.position.y,
                                  s2.position.z - s1.position.z);

        // Distance between spheres
        float dist2 = diff.x * diff.x + diff.y * diff.y + diff.z * diff.z;
        float min_dist = s1.radius + s2.radius;

        // If they are colliding
        if (dist2 < min_dist * min_dist) {
            float dist = sqrtf(dist2);
            float overlap = 0.5f * (min_dist - dist);

            // Normalize the difference vector
            diff.x /= dist;
            diff.y /= dist;
            diff.z /= dist;

            // Move the spheres apart based on the overlap
            s1.position.x -= overlap * diff.x;
            s1.position.y -= overlap * diff.y;
            s1.position.z -= overlap * diff.z;

            s2.position.x += overlap * diff.x;
            s2.position.y += overlap * diff.y;
            s2.position.z += overlap * diff.z;

            // Swap velocities in the direction of collision
            float3 velocity_diff = make_float3(s2.velocity.x - s1.velocity.x,
                                               s2.velocity.y - s1.velocity.y,
                                               s2.velocity.z - s1.velocity.z);
            float velocity_dot = velocity_diff.x * diff.x + velocity_diff.y * diff.y + velocity_diff.z * diff.z;

            if (velocity_dot > 0.0f) {
                s1.velocity.x += velocity_dot * diff.x;
                s1.velocity.y += velocity_dot * diff.y;
                s1.velocity.z += velocity_dot * diff.z;

                s2.velocity.x -= velocity_dot * diff.x;
                s2.velocity.y -= velocity_dot * diff.y;
                s2.velocity.z -= velocity_dot * diff.z;
            }
        }
    }
}

// Utility function to check for CUDA errors
hipError_t NBodySimulation::checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        exit(-1);
    }
    return result;
}

// Constructor
NBodySimulation::NBodySimulation(int num_spheres) : num_spheres(num_spheres) {
    h_spheres.resize(num_spheres);
    initializeSpheres();

    checkCuda(hipMalloc(&d_spheres, num_spheres * sizeof(Sphere)));

    checkCuda(hipMemcpy(d_spheres, h_spheres.data(), num_spheres * sizeof(Sphere), hipMemcpyHostToDevice));
}

// Destructor
NBodySimulation::~NBodySimulation() {
    checkCuda(hipFree(d_spheres));
}

void NBodySimulation::initializeSpheres() {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> pos_dist(0.0f, BOX_SIZE);
    std::uniform_real_distribution<float> vel_dist(-5.0f, 5.0f);

    auto generate_sphere = [&]() {
        return Sphere{
            make_float3(pos_dist(gen), pos_dist(gen), pos_dist(gen)),
            make_float3(vel_dist(gen), vel_dist(gen), vel_dist(gen)),
            1.0f
        };
    };

    std::generate(h_spheres.begin(), h_spheres.end(), generate_sphere);
}

void NBodySimulation::update() {
    // Define the number of threads and blocks
    int threadsPerBlock = 256;
    int blocksPerGrid = (num_spheres + threadsPerBlock - 1) / threadsPerBlock;

    update_positions<<<blocksPerGrid, threadsPerBlock>>>(d_spheres, num_spheres, TIME_STEP);
    checkCuda(hipDeviceSynchronize());

    resolve_collisions<<<blocksPerGrid, threadsPerBlock>>>(d_spheres, num_spheres);
    checkCuda(hipDeviceSynchronize());

    checkCuda(hipMemcpy(h_spheres.data(), d_spheres, num_spheres * sizeof(Sphere), hipMemcpyDeviceToHost));
}

// Accessor for spheres
const std::vector<Sphere>& NBodySimulation::getSpheres() const {
    return h_spheres;
}